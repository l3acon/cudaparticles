#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define BIN_SIZE cutoff
#define NUM_THREADS 256
#define MPPB 8		// MAX_PARTICLES_PER_BIN was too long

extern double size;

// __device__ functions are callable from DEVICE ONLY	
__device__ int2 calBin( float x, float y)
{
	int2 binPos;
	binPos.x = floor(x/BIN_SIZE);
	binPos.y = floor(y/BIN_SIZE);
	return binPos;
}

__device__ void applyForce(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;
}


//	this is a kernel
//__global__ void initBins(int n)
//{
//	//	number of particles in each bin
//	//	initialilzed to zero inbetween each step
//	int* binCounters = (int*) a;
//
//	//	indicies of each particle in a particular bin
//	//	only has room for MAX_PARTIVLES_PER_BIN
//	particle_t *particlesInBin = (particle_t*) binCounters[n];
//	
//	//	the size of numParticlesInBin is:
//	//	NUM_PARTICLES * MPPB
//	int* unused = (int*) numParticlesInBin[n*MPPB]; 
//}

__global__ void clearBins(int n)
{
	extern __shared__ int binCounters[];
	extern __shared__ particle_t* particlesInBin[];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;
	binCounters[tid] = 0;
	for(int i = 0; i < MPPB; ++i)
		particlesInBin[tid*MPPB+i] = 0;	//	not using NULL, maybe should
}


//	this is called on all BINS
__global__ void binCollide(int side, int n)
{
	extern __shared__ int binCounters[];
	extern __shared__ particle_t* particlesInBin[];


  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  	
	if(tid >= n) 
		return;
	
	int pITB = binCounters[tid];

	// for all particles in this bin
	for(int j = 0; j < pITB; ++j)
	{
		// reset the acceleration 
		particlesInBin[j]->ax = particlesInBin[j]->ay = 0;

		//
		// intra-bin forces
		for(int k = 0; k < binCounters[tid]; ++k)
			applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[tid*MPPB + k]);

		//	
		// left bin
		if(tid%side != 0) // if i is not leftmost in row
			for(int k = 0; k < binCounters[tid-1]; ++k)
				applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid-1)*MPPB + k] );

		//	
		// right bin
		if(tid%side != side-1) // if i is not rightmost in row
			for(int k = 0; k < binCounters[tid+1]; ++k)
				applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid+1)*MPPB + k] );

		//
		// up bins
		if(tid >= side) //	all but the first row 
		{
			if(tid%side > 0) // make sure we're not leftmost in row
				for(int k = 0; k < binCounters[tid-side-1]; ++k)
					applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid-side-1)*MPPB+k]);

			for(int k = 0; k < binCounters[tid-side]; ++k)
					applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid-side)*MPPB+k]);
		
			if(tid%side < side-1) // make sure we're not rightmost in row
				for(int k = 0; k < binCounters[tid-side+1]; ++k)
					applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid-side+1)*MPPB+k]);
		}
	
		//		
		// down bins
		if(tid <= side*side - side-1) //	not sure if +1 or -1
		{
			if(tid%side > 0) // make sure we're not leftmost in row
				for(int k = 0; k < binCounters[tid+side-1]; ++k)
					applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid+side-1)*MPPB+k]);
	 
			for(int k = 0; k < binCounters[tid+side]; ++k)
				applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid+side)*MPPB+k]);
		
			if(tid%side < side-1) // make sure we're not rightmost in row
				for(int k = 0; k < binCounters[tid+side+1]; ++k)
					applyForce( *particlesInBin[tid*MPPB + j], *particlesInBin[(tid+side+1)*MPPB+k]); 
		}
	}
}

__global__ void moveParticles (particle_t * particles, int n, double size)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
	if(tid >= n) 
		return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }
}

//	called for all PARTICLES
__global__ void updateBins( particle_t *particles, int side, int n )
{
	extern __shared__ int binCounters[];
	extern __shared__ particle_t* particlesInBin[];


	//	threadIdx * blockIdx is the particle, 
	//	blockDim is usually just 1 (I hope)
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
	if(tid >= n) 
		return;
	
	int2 bi = calBin(particles[tid].x, particles[tid].y);	//	calculate bin index
	atomicAdd(&binCounters[bi.x+side*bi.y],1);
	if(binCounters[bi.x*bi.y] >= MPPB || particlesInBin[binCounters[bi.x+side*bi.y]] != 0)
		return;	//	probably do something else for this error

	//	add our particle index to the bin	
	//	NOTE: this whole thing might need to be atomic?
	particlesInBin[binCounters[bi.x+side*bi.y]] = &particles[tid];
}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
	printf( "-s <filename> to specify the summary output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    char *sumname = read_string( argc, argv, "-s", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    FILE *fsum = sumname ? fopen(sumname,"a") : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
		//	need to get number of bins somehow?
		int side = set_size( n );

		// GPU particle data structure
    particle_t * d_particles;

		//	//	number of particles in each bin
		//	initialilzed to zero inbetween each step
		int h_binCounters [n];
		int * binCounters;
		double copy_time = read_timer( );
		//	indicies of each particle in a particular bin
		//	only has room for MAX_PARTIVLES_PER_BIN
		particle_t h_particlesInBin[n*MPPB] ;
		particle_t *particlesInBin;

		//	allocate device memory
    hipMalloc( (void **) &d_particles, n * sizeof(particle_t));
		hipMalloc( (int **) &binCounters, n*sizeof(int));
		hipMalloc( (void **) &particlesInBin, n*MPPB*sizeof(particle_t*));

		//	only the particles need to be copied to the device
		hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int pblks = (n + NUM_THREADS - 1) / NUM_THREADS;
		int blks = ( + NUM_THREADS -1) / NUM_THREADS;
		
		//	for all particles
		//initBins<<< pblks, NUM_THREADS >>> (n);
		updateBins<<< pblks, NUM_THREADS >>>(d_particles, side, n);
    copy_time = read_timer( ) - copy_time;
		for( int step = 0; step < NSTEPS; step++ )
		{
			//	for all bins
			binCollide<<< blks, NUM_THREADS >>>( side, n);
			hipDeviceSynchronize();	

			//	for all particles
			moveParticles <<< pblks, NUM_THREADS >>> (d_particles, n, size);
			hipDeviceSynchronize();
			
			//	for all bins
			clearBins<<<blks, NUM_THREADS >>>(n);

			hipDeviceSynchronize();
			updateBins<<< blks, NUM_THREADS >>> (d_particles, side, n);
     	
			//
      //  save if necessary
      //
      if( fsave && (step%SAVEFREQ) == 0 ) 
			{
	    	// Copy the particles back to the CPU
      	hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
       	save( fsave, n, particles);
			}
		}

    hipDeviceSynchronize();
    double simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

    if (fsum)
	fprintf(fsum,"%d %lf \n",n,simulation_time);

    if (fsum)
	fclose( fsum );    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}





